//!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
//
// MODULE linalg
//
//!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
// This module contains wrappers for linear algebra subroutines

#include <iostream>
#include <stdio.h>
#include "hipsolver.h"
#include "global.h"
#include "utils.h"
#include "matrix.h"
#include "linalg.h"

//!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
// Host-callable functions
//!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

////////////////////////////////////////////////////////////////////////

__host__ void solve_geneigval(Matrix &QHQ_d, const Matrix &S_d, prec_typ *eigv_d, const int &n)

////////////////////////////////////////////////////////////////////////
// Solves general eigenvalue problem. Arrays QHQ_d, S_d, and eigv_d must
// be allocated on device before calling

{
// CUSOLVER handles and initialization
   hipsolverHandle_t gep;
   hipsolverEigType_t ITYP=HIPSOLVER_EIG_TYPE_1;
   hipsolverEigMode_t JOB=HIPSOLVER_EIG_MODE_VECTOR;
   hipblasFillMode_t UPLO=HIPBLAS_FILL_MODE_LOWER;
   hipsolverStatus_t stat;
   stat = hipsolverDnCreate(&gep);
   if (stat!=HIPSOLVER_STATUS_SUCCESS){
      throwERROR("error with hipsolverDnCreate()");
   }
   prec_typ *WORK;
   int *INFO;
   int LWORK;

// Allocate workspace
#ifdef USE_DOUBLES
   stat = hipsolverDnDsygvd_bufferSize(gep,ITYP,JOB,UPLO,n,&QHQ_d.mat[0],n,&S_d.mat[0],n,&eigv_d[0],&LWORK);
#else
   stat = hipsolverDnSsygvd_bufferSize(gep,ITYP,JOB,UPLO,n,&QHQ_d.mat[0],n,&S_d.mat[0],n,&eigv_d[0],&LWORK);
#endif
   if (stat!=HIPSOLVER_STATUS_SUCCESS){
      throwERROR("error with cusolverDnXsygvd_bufferSize()");
   }
   hipMalloc(&WORK,LWORK*sizeof(prec_typ));
   hipMalloc(&INFO,sizeof(int));

// Solve generalized eigenvalue problem
#ifdef USE_DOUBLES
   stat = hipsolverDnDsygvd(gep,ITYP,JOB,UPLO,n,&QHQ_d.mat[0],n,&S_d.mat[0],n,&eigv_d[0],&WORK[0],LWORK,INFO);
#else
   stat = hipsolverDnSsygvd(gep,ITYP,JOB,UPLO,n,&QHQ_d.mat[0],n,&S_d.mat[0],n,&eigv_d[0],&WORK[0],LWORK,INFO);
#endif
   if (stat!=HIPSOLVER_STATUS_SUCCESS){
      throwERROR("error with cusolverDnXsygvd()");
   }

// Cleanup
   hipFree(WORK);
   hipFree(INFO);
   hipsolverDnDestroy(gep);
}

////////////////////////////////////////////////////////////////////////

__host__ void solve_linsys(Matrix &BB_d, Matrix &bjk_d, prec_typ *WORK, int *IPV, int *INFO, const int &rk, const int &n)

////////////////////////////////////////////////////////////////////////
// Solves system of linear equations. All arrays must be allocated on 
// device before calling

{

// Set up CUSOLVER
   hipsolverHandle_t ls;
   hipblasOperation_t trans = HIPBLAS_OP_N;
   hipsolverStatus_t stat;
   stat = hipsolverDnCreate(&ls);
   if (stat!=HIPSOLVER_STATUS_SUCCESS){
      throwERROR("error with hipsolverDnCreate()");
   }

// Solve linear system BB*cjk = bjk with DGETRF + DGETRS
#ifdef USE_DOUBLES
   stat = hipsolverDnDgetrf(ls,rk,rk,&BB_d.mat[0],rk,&WORK[0],&IPV[0],INFO);
#else
   stat = hipsolverDnSgetrf(ls,rk,rk,&BB_d.mat[0],rk,&WORK[0],&IPV[0],INFO);
#endif
   if (stat!=HIPSOLVER_STATUS_SUCCESS){
      throwERROR("error with hipsolverDnXgetrf()");
   }

#ifdef USE_DOUBLES
   stat = hipsolverDnDgetrs(ls,trans,rk,n,&BB_d.mat[0],rk,&IPV[0],&bjk_d.mat[0],rk,INFO);
#else
   stat = hipsolverDnSgetrs(ls,trans,rk,n,&BB_d.mat[0],rk,&IPV[0],&bjk_d.mat[0],rk,INFO);
#endif
   if (stat!=HIPSOLVER_STATUS_SUCCESS){
      throwERROR("error with hipsolverDnXgetrs()");
   }

// Clean up
   hipsolverDnDestroy(ls);
}

//!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
